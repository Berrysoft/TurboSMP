#include "hip/hip_runtime.h"
#include <cassert>
#include <chrono>
#include <cmath>
#include <iostream>
#include <random>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#ifdef __INTELLISENSE__
    #define CUDA_KERNEL(...)
    #define __HIPCC__
#else
    #define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

#include <hip/hip_runtime.h>

using namespace std;

// Thread 0 can assume the result is calculated successfully.
__device__ void dot(const size_t n, const float* x, const float* y, float* pres)
{
    __shared__ float temp[1024];
    assert(n < 1024);
    const uint32_t id = threadIdx.x;
    if (id < n)
    {
        temp[id] += x[id] * y[id];
    }
    __syncthreads();
    // TODO: optimize sum
    if (id == 0)
    {
        float res = 0.0;
        for (size_t i = 0; i < n; i++)
        {
            res += temp[i];
        }
        *pres = res;
    }
}

__device__ void combine(
    const size_t nw,
    const float* A,
    const float* cx,
    const float t, // 时间，属于 tlist 下标
    float* A_vec, // nw
    float* c_vec // nw
)
{
}

// TODO: t is ordered
__device__ void real_time(
    const size_t n,
    const size_t nl,
    float* pt, // n
    const float* tlist // nl
)
{
    const uint32_t id = threadIdx.x;
    if (id < n)
    {
        float t = pt[id];
        // TODO: bisect
        size_t i = 0;
        for (; i < nl; i++)
        {
            if (tlist[i] > t) break;
        }
        assert(i > 0);
        assert(i < nl);
        pt[id] = i - 1 + (t - tlist[i - 1]) / (tlist[i] - tlist[i - 1]);
    }
}

__device__ void lc(const size_t n, float* t)
{
}

__device__ void move1(
    const size_t nw,
    const size_t nl,
    const float* A_vec,
    const float* c_vec,
    const float* z,
    const int step,
    const float mus,
    const float sig2s,
    float* pdelta_nu, // x1
    float* pbeta // x1
)
{
    const uint32_t id = threadIdx.x;
    float beta_under;
    dot(nw, A_vec, c_vec, &beta_under);
    float temp;
    dot(nw, z, c_vec, &temp);
    if (id == 0)
    {
        float fsig2s = step * sig2s;
        beta_under = 1 + fsig2s * beta_under;
        float beta = fsig2s / beta_under;
        float delta_nu = 0.5 * (beta * powf(temp + mus / sig2s, 2.0f) - powf(mus, 2.0f) / sig2s);
        delta_nu -= 0.5 * logf(beta_under);
        *pdelta_nu = delta_nu;
        *pbeta = beta;
    }
}

__device__ void move2(
    const size_t nw,
    const size_t nl,
    const float* A_vec,
    const float* c_vec,
    const int step,
    const float mus,
    const float* A,
    const float beta,
    float* delta_cx, // nw x nl
    float* delta_z // nw
)
{
}

// return delta_nu
__device__ void move(
    const size_t nw,
    const size_t nl,
    const float* A_vec,
    const float* c_vec,
    const float* z,
    const int step,
    const float mus,
    const float sig2s,
    const float* A,
    float* delta_nu,
    float* delta_cx,
    float* delta_z)
{
    __shared__ float beta;
    move1(nw, nl, A_vec, c_vec, z, step, mus, sig2s, delta_nu, &beta);
    __syncthreads();
    move2(nw, nl, A_vec, c_vec, step, mus, A, beta, delta_cx, delta_z);
}

static constexpr size_t TRIALS = 2000;

__global__ void flow(
    const size_t nw,
    const size_t nl,
    const float* cx, // nw x nl, Cov^-1 * A, 详见 FBMP
    const float* tlist, // nl
    const float* z, // nw, residue waveform
    const float mus, // spe波形的平均幅值
    const float sig2s, // spe波形幅值方差
    const float* A, // nw x nl
    const float* p_cha, // nl
    const float mu_t, // LucyDDM 的估算 PE 数
    float* s0_history, // TRIALS, ||s||_0
    float* delta_nu_history, // TRIALS, Δν
    float* t0_history, // TRIALS, t_0
    float* A_vec,
    float* c_vec)
{
    // 波形编号，一个 block 一个波形
    const uint32_t waveform_id = blockIdx.x;
    // 并行用
    const uint32_t thread_id = threadIdx.x;
}

int main()
{
    return 0;
}
